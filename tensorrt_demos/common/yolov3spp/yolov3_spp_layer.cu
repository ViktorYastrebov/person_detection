#include "hip/hip_runtime.h"
#include "yolov3_spp_layer.h"

using namespace YoloV3SPP;

namespace nvinfer1
{
    YoloV3SPPLayerPlugin::YoloV3SPPLayerPlugin()
    {
        mClassCount = CLASS_NUM;
        mYoloKernel.clear();
        mYoloKernel.push_back(yolo1);
        mYoloKernel.push_back(yolo2);
        mYoloKernel.push_back(yolo3);

        mKernelCount = mYoloKernel.size();
    }
    
    YoloV3SPPLayerPlugin::~YoloV3SPPLayerPlugin()
    {
    }

    // create the plugin at runtime from a byte stream
    YoloV3SPPLayerPlugin::YoloV3SPPLayerPlugin(const void* data, size_t length)
    {
        using namespace Tn;
        const char *d = reinterpret_cast<const char *>(data), *a = d;
        read(d, mClassCount);
        read(d, mThreadCount);
        read(d, mKernelCount);
        mYoloKernel.resize(mKernelCount);
        auto kernelSize = mKernelCount*sizeof(YoloKernel);
        memcpy(mYoloKernel.data(),d,kernelSize);
        d += kernelSize;

        assert(d == a + length);
    }

    void YoloV3SPPLayerPlugin::serialize(void* buffer) const
    {
        using namespace Tn;
        char* d = static_cast<char*>(buffer), *a = d;
        write(d, mClassCount);
        write(d, mThreadCount);
        write(d, mKernelCount);
        auto kernelSize = mKernelCount*sizeof(YoloKernel);
        memcpy(d,mYoloKernel.data(),kernelSize);
        d += kernelSize;

        assert(d == a + getSerializationSize());
    }
    
    size_t YoloV3SPPLayerPlugin::getSerializationSize() const
    {  
        return sizeof(mClassCount) + sizeof(mThreadCount) + sizeof(mKernelCount)  + sizeof(YoloV3SPP::YoloKernel) * mYoloKernel.size();
    }

    int YoloV3SPPLayerPlugin::initialize()
    { 
        return 0;
    }
    
    Dims YoloV3SPPLayerPlugin::getOutputDimensions(int index, const Dims* inputs, int nbInputDims)
    {
        //output the result to channel
        int totalsize = MAX_OUTPUT_BBOX_COUNT * sizeof(Detection) / sizeof(float);

        return Dims3(totalsize + 1, 1, 1);
    }

    // Set plugin namespace
    void YoloV3SPPLayerPlugin::setPluginNamespace(const char* pluginNamespace)
    {
        mPluginNamespace = pluginNamespace;
    }

    const char* YoloV3SPPLayerPlugin::getPluginNamespace() const
    {
        return mPluginNamespace;
    }

    // Return the DataType of the plugin output at the requested index
    DataType YoloV3SPPLayerPlugin::getOutputDataType(int index, const nvinfer1::DataType* inputTypes, int nbInputs) const
    {
        return DataType::kFLOAT;
    }

    // Return true if output tensor is broadcast across a batch.
    bool YoloV3SPPLayerPlugin::isOutputBroadcastAcrossBatch(int outputIndex, const bool* inputIsBroadcasted, int nbInputs) const
    {
        return false;
    }

    // Return true if plugin can use input that is broadcast across batch without replication.
    bool YoloV3SPPLayerPlugin::canBroadcastInputAcrossBatch(int inputIndex) const
    {
        return false;
    }

    void YoloV3SPPLayerPlugin::configurePlugin(const PluginTensorDesc* in, int nbInput, const PluginTensorDesc* out, int nbOutput)
    {
    }

    // Attach the plugin object to an execution context and grant the plugin the access to some context resource.
    void YoloV3SPPLayerPlugin::attachToContext(cudnnContext* cudnnContext, cublasContext* cublasContext, IGpuAllocator* gpuAllocator)
    {
    }

    // Detach the plugin object from its execution context.
    void YoloV3SPPLayerPlugin::detachFromContext() {}

    const char* YoloV3SPPLayerPlugin::getPluginType() const
    {
        //return "YoloLayer_TRT";
        return "YoloV3SPPLayer_TRT";
    }

    const char* YoloV3SPPLayerPlugin::getPluginVersion() const
    {
        return "1";
    }

    void YoloV3SPPLayerPlugin::destroy()
    {
        delete this;
    }

    // Clone the plugin
    IPluginV2IOExt* YoloV3SPPLayerPlugin::clone() const
    {
        YoloV3SPPLayerPlugin *p = new YoloV3SPPLayerPlugin();
        p->setPluginNamespace(mPluginNamespace);
        return p;
    }

    __device__ float Logist(float data){ return 1.0f / (1.0f + expf(-data)); };

    __global__ void CalDetection(const float *input, float *output,int noElements, 
            int yoloWidth,int yoloHeight,const float anchors[CHECK_COUNT*2],int classes,int outputElem) {
 
        int idx = threadIdx.x + blockDim.x * blockIdx.x;
        if (idx >= noElements) return;

        int total_grid = yoloWidth * yoloHeight;
        int bnIdx = idx / total_grid;
        idx = idx - total_grid*bnIdx;
        int info_len_i = 5 + classes;
        const float* curInput = input + bnIdx * (info_len_i * total_grid * CHECK_COUNT);

        for (int k = 0; k < 3; ++k) {
            int class_id = 0;
            float max_cls_prob = 0.0;
            for (int i = 5; i < info_len_i; ++i) {
                float p = Logist(curInput[idx + k * info_len_i * total_grid + i * total_grid]);
                if (p > max_cls_prob) {
                    max_cls_prob = p;
                    class_id = i - 5;
                }
            }
            float box_prob = Logist(curInput[idx + k * info_len_i * total_grid + 4 * total_grid]);
            if (max_cls_prob < IGNORE_THRESH || box_prob < IGNORE_THRESH) continue;

            float *res_count = output + bnIdx*outputElem;
            int count = (int)atomicAdd(res_count, 1);
            if (count >= MAX_OUTPUT_BBOX_COUNT) return;
            char* data = (char * )res_count + sizeof(float) + count*sizeof(Detection);
            Detection* det =  (Detection*)(data);

            int row = idx / yoloWidth;
            int col = idx % yoloWidth;

            //Location
            det->bbox[0] = (col + Logist(curInput[idx + k * info_len_i * total_grid + 0 * total_grid])) * INPUT_W / yoloWidth;
            det->bbox[1] = (row + Logist(curInput[idx + k * info_len_i * total_grid + 1 * total_grid])) * INPUT_H / yoloHeight;
            det->bbox[2] = expf(curInput[idx + k * info_len_i * total_grid + 2 * total_grid]) * anchors[2*k];
            det->bbox[3] = expf(curInput[idx + k * info_len_i * total_grid + 3 * total_grid]) * anchors[2*k + 1];
            det->det_confidence = box_prob;
            det->class_id = class_id;
            det->class_confidence = max_cls_prob;
        }
    }

    void YoloV3SPPLayerPlugin::forwardGpu(const float *const * inputs, float* output, hipStream_t stream, int batchSize) {
        void* devAnchor;
        size_t AnchorLen = sizeof(float)* CHECK_COUNT*2;
        CUDA_CHECK(hipMalloc(&devAnchor,AnchorLen));

        int outputElem = 1 + MAX_OUTPUT_BBOX_COUNT * sizeof(Detection) / sizeof(float);

        for(int idx = 0 ; idx < batchSize; ++idx) {
            CUDA_CHECK(hipMemset(output + idx*outputElem, 0, sizeof(float)));
        }
        int numElem = 0;
        for (unsigned int i = 0;i< mYoloKernel.size();++i)
        {
            const auto& yolo = mYoloKernel[i];
            numElem = yolo.width*yolo.height*batchSize;
            if (numElem < mThreadCount)
                mThreadCount = numElem;
            CUDA_CHECK(hipMemcpy(devAnchor, yolo.anchors, AnchorLen, hipMemcpyHostToDevice));
            CalDetection<<< (yolo.width*yolo.height*batchSize + mThreadCount - 1) / mThreadCount, mThreadCount>>>
                (inputs[i],output, numElem, yolo.width, yolo.height, (float *)devAnchor, mClassCount ,outputElem);
        }

        CUDA_CHECK(hipFree(devAnchor));
    }


    int YoloV3SPPLayerPlugin::enqueue(int batchSize, const void*const * inputs, void** outputs, void* workspace, hipStream_t stream)
    {
        //assert(batchSize == 1);
        //GPU
        //CUDA_CHECK(hipStreamSynchronize(stream));
        forwardGpu((const float *const *)inputs, (float*)outputs[0], stream, batchSize);

        return 0;
    }

    PluginFieldCollection YoloV3SPPPluginCreator::mFC{};
    std::vector<PluginField> YoloV3SPPPluginCreator::mPluginAttributes;

    YoloV3SPPPluginCreator::YoloV3SPPPluginCreator()
    {
        mPluginAttributes.clear();

        mFC.nbFields = mPluginAttributes.size();
        mFC.fields = mPluginAttributes.data();
    }

    const char* YoloV3SPPPluginCreator::getPluginName() const
    {
        //return "YoloLayer_TRT";
        return "YoloV3SPPLayer_TRT";
    }

    const char* YoloV3SPPPluginCreator::getPluginVersion() const
    {
            return "1";
    }

    const PluginFieldCollection* YoloV3SPPPluginCreator::getFieldNames()
    {
            return &mFC;
    }

    IPluginV2IOExt* YoloV3SPPPluginCreator::createPlugin(const char* name, const PluginFieldCollection* fc)
    {
        YoloV3SPPLayerPlugin* obj = new YoloV3SPPLayerPlugin();
        obj->setPluginNamespace(mNamespace.c_str());
        return obj;
    }

    IPluginV2IOExt* YoloV3SPPPluginCreator::deserializePlugin(const char* name, const void* serialData, size_t serialLength)
    {
        // This object will be deleted when the network is destroyed, which will
        // call MishPlugin::destroy()
        YoloV3SPPLayerPlugin* obj = new YoloV3SPPLayerPlugin(serialData, serialLength);
        obj->setPluginNamespace(mNamespace.c_str());
        return obj;
    }

}
