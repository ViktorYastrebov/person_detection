#include "hip/hip_runtime.h"
#include <assert.h>
#include "yolov5_layer.h"
#include "utils.h"

using namespace YoloV5;

namespace nvinfer1
{
    YoloV5LayerPlugin::YoloV5LayerPlugin()
    {
        mClassCount = CLASS_NUM;
        mYoloKernel.clear();
        mYoloKernel.push_back(yolo1);
        mYoloKernel.push_back(yolo2);
        mYoloKernel.push_back(yolo3);

        mKernelCount = mYoloKernel.size();

        CUDA_CHECK(hipHostMalloc(&mAnchor, mKernelCount * sizeof(void*)));
        size_t AnchorLen = sizeof(float)* CHECK_COUNT*2;
        for(int ii = 0; ii < mKernelCount; ii ++)
        {
            CUDA_CHECK(hipMalloc(&mAnchor[ii],AnchorLen));
            const auto& yolo = mYoloKernel[ii];
            CUDA_CHECK(hipMemcpy(mAnchor[ii], yolo.anchors, AnchorLen, hipMemcpyHostToDevice));
        }
    }
    
    YoloV5LayerPlugin::~YoloV5LayerPlugin()
    {
    }

    // create the plugin at runtime from a byte stream
    YoloV5LayerPlugin::YoloV5LayerPlugin(const void* data, size_t length)
    {
        using namespace Tn;
        const char *d = reinterpret_cast<const char *>(data), *a = d;
        read(d, mClassCount);
        read(d, mThreadCount);
        read(d, mKernelCount);
        mYoloKernel.resize(mKernelCount);
        auto kernelSize = mKernelCount*sizeof(YoloKernel);
        memcpy(mYoloKernel.data(),d,kernelSize);
        d += kernelSize;

        CUDA_CHECK(hipHostMalloc(&mAnchor, mKernelCount * sizeof(void*)));
        size_t AnchorLen = sizeof(float)* CHECK_COUNT*2;
        for(int ii = 0; ii < mKernelCount; ii ++)
        {
            CUDA_CHECK(hipMalloc(&mAnchor[ii],AnchorLen));
            const auto& yolo = mYoloKernel[ii];
            CUDA_CHECK(hipMemcpy(mAnchor[ii], yolo.anchors, AnchorLen, hipMemcpyHostToDevice));
        }

        assert(d == a + length);
    }

    void YoloV5LayerPlugin::serialize(void* buffer) const
    {
        using namespace Tn;
        char* d = static_cast<char*>(buffer), *a = d;
        write(d, mClassCount);
        write(d, mThreadCount);
        write(d, mKernelCount);
        auto kernelSize = mKernelCount*sizeof(YoloKernel);
        memcpy(d,mYoloKernel.data(),kernelSize);
        d += kernelSize;

        assert(d == a + getSerializationSize());
    }
    
    size_t YoloV5LayerPlugin::getSerializationSize() const
    {  
        return sizeof(mClassCount) + sizeof(mThreadCount) + sizeof(mKernelCount)  + sizeof(YoloV5::YoloKernel) * mYoloKernel.size();
    }

    int YoloV5LayerPlugin::initialize()
    { 
        return 0;
    }
    
    Dims YoloV5LayerPlugin::getOutputDimensions(int index, const Dims* inputs, int nbInputDims)
    {
        //output the result to channel
        int totalsize = MAX_OUTPUT_BBOX_COUNT * sizeof(Detection) / sizeof(float);

        return Dims3(totalsize + 1, 1, 1);
    }

    // Set plugin namespace
    void YoloV5LayerPlugin::setPluginNamespace(const char* pluginNamespace)
    {
        mPluginNamespace = pluginNamespace;
    }

    const char* YoloV5LayerPlugin::getPluginNamespace() const
    {
        return mPluginNamespace;
    }

    // Return the DataType of the plugin output at the requested index
    DataType YoloV5LayerPlugin::getOutputDataType(int index, const nvinfer1::DataType* inputTypes, int nbInputs) const
    {
        return DataType::kFLOAT;
    }

    // Return true if output tensor is broadcast across a batch.
    bool YoloV5LayerPlugin::isOutputBroadcastAcrossBatch(int outputIndex, const bool* inputIsBroadcasted, int nbInputs) const
    {
        return false;
    }

    // Return true if plugin can use input that is broadcast across batch without replication.
    bool YoloV5LayerPlugin::canBroadcastInputAcrossBatch(int inputIndex) const
    {
        return false;
    }

    void YoloV5LayerPlugin::configurePlugin(const PluginTensorDesc* in, int nbInput, const PluginTensorDesc* out, int nbOutput)
    {
    }

    // Attach the plugin object to an execution context and grant the plugin the access to some context resource.
    void YoloV5LayerPlugin::attachToContext(cudnnContext* cudnnContext, cublasContext* cublasContext, IGpuAllocator* gpuAllocator)
    {
    }

    // Detach the plugin object from its execution context.
    void YoloV5LayerPlugin::detachFromContext() {}

    const char* YoloV5LayerPlugin::getPluginType() const
    {
        return "YoloV5Layer_TRT";
    }

    const char* YoloV5LayerPlugin::getPluginVersion() const
    {
        return "1";
    }

    void YoloV5LayerPlugin::destroy()
    {
        delete this;
    }

    // Clone the plugin
    IPluginV2IOExt* YoloV5LayerPlugin::clone() const
    {
        YoloV5LayerPlugin *p = new YoloV5LayerPlugin();
        p->setPluginNamespace(mPluginNamespace);
        return p;
    }

    __device__ float Logist(float data){ return 1.0f / (1.0f + expf(-data)); };

    __global__ void CalDetection(const float *input, float *output,int noElements, 
            int yoloWidth,int yoloHeight,const float anchors[CHECK_COUNT*2],int classes,int outputElem) {
 
        int idx = threadIdx.x + blockDim.x * blockIdx.x;
        if (idx >= noElements) return;

        int total_grid = yoloWidth * yoloHeight;
        int bnIdx = idx / total_grid;
        idx = idx - total_grid*bnIdx;
        int info_len_i = 5 + classes;
        const float* curInput = input + bnIdx * (info_len_i * total_grid * CHECK_COUNT);

        for (int k = 0; k < 3; ++k) {
            float box_prob = Logist(curInput[idx + k * info_len_i * total_grid + 4 * total_grid]);
            if (box_prob < IGNORE_THRESH) continue;
            int class_id = 0;
            float max_cls_prob = 0.0;
            for (int i = 5; i < info_len_i; ++i) {
                float p = Logist(curInput[idx + k * info_len_i * total_grid + i * total_grid]);
                if (p > max_cls_prob) {
                    max_cls_prob = p;
                    class_id = i - 5;
                }
            }
            float *res_count = output + bnIdx*outputElem;
            int count = (int)atomicAdd(res_count, 1);
            if (count >= MAX_OUTPUT_BBOX_COUNT) return;
            char* data = (char *)res_count + sizeof(float) + count * sizeof(Detection);
            YoloV5::Detection* det =  (YoloV5::Detection*)(data);

            int row = idx / yoloWidth;
            int col = idx % yoloWidth;

            //Location
            det->bbox[0] = (col - 0.5f + 2.0f * Logist(curInput[idx + k * info_len_i * total_grid + 0 * total_grid])) * INPUT_W / yoloWidth;
            det->bbox[1] = (row - 0.5f + 2.0f * Logist(curInput[idx + k * info_len_i * total_grid + 1 * total_grid])) * INPUT_H / yoloHeight;
            det->bbox[2] = 2.0f * Logist(curInput[idx + k * info_len_i * total_grid + 2 * total_grid]);
            det->bbox[2] = det->bbox[2] * det->bbox[2] * anchors[2*k];
            det->bbox[3] = 2.0f * Logist(curInput[idx + k * info_len_i * total_grid + 3 * total_grid]);
            det->bbox[3] = det->bbox[3] * det->bbox[3] * anchors[2*k + 1];
            det->conf = box_prob * max_cls_prob;
            det->class_id = class_id;
        }
    }

    void YoloV5LayerPlugin::forwardGpu(const float *const * inputs, float* output, hipStream_t stream, int batchSize) {

        int outputElem = 1 + MAX_OUTPUT_BBOX_COUNT * sizeof(Detection) / sizeof(float);

        for(int idx = 0 ; idx < batchSize; ++idx) {
            CUDA_CHECK(hipMemset(output + idx*outputElem, 0, sizeof(float)));
        }
        int numElem = 0;
        for (unsigned int i = 0; i < mYoloKernel.size(); ++i)
        {
            const auto& yolo = mYoloKernel[i];
            numElem = yolo.width*yolo.height*batchSize;
            if (numElem < mThreadCount)
                mThreadCount = numElem;
            CalDetection<<< (yolo.width*yolo.height*batchSize + mThreadCount - 1) / mThreadCount, mThreadCount>>>
                (inputs[i], output, numElem, yolo.width, yolo.height, (float *)mAnchor[i], mClassCount, outputElem);
        }

    }


    int YoloV5LayerPlugin::enqueue(int batchSize, const void*const * inputs, void** outputs, void* workspace, hipStream_t stream)
    {
        forwardGpu((const float *const *)inputs, (float*)outputs[0], stream, batchSize);
        return 0;
    }

    PluginFieldCollection YoloV5PluginCreator::mFC{};
    std::vector<PluginField> YoloV5PluginCreator::mPluginAttributes;

    YoloV5PluginCreator::YoloV5PluginCreator()
    {
        mPluginAttributes.clear();

        mFC.nbFields = mPluginAttributes.size();
        mFC.fields = mPluginAttributes.data();
    }

    const char* YoloV5PluginCreator::getPluginName() const
    {
            return "YoloV5Layer_TRT";
    }

    const char* YoloV5PluginCreator::getPluginVersion() const
    {
            return "1";
    }

    const PluginFieldCollection* YoloV5PluginCreator::getFieldNames()
    {
            return &mFC;
    }

    IPluginV2IOExt* YoloV5PluginCreator::createPlugin(const char* name, const PluginFieldCollection* fc)
    {
        YoloV5LayerPlugin* obj = new YoloV5LayerPlugin();
        obj->setPluginNamespace(mNamespace.c_str());
        return obj;
    }

    IPluginV2IOExt* YoloV5PluginCreator::deserializePlugin(const char* name, const void* serialData, size_t serialLength)
    {
        // This object will be deleted when the network is destroyed, which will
        // call MishPlugin::destroy()
        YoloV5LayerPlugin* obj = new YoloV5LayerPlugin(serialData, serialLength);
        obj->setPluginNamespace(mNamespace.c_str());
        return obj;
    }
    REGISTER_TENSORRT_PLUGIN(YoloV5PluginCreator);
}

